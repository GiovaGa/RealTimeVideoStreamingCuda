#include "hip/hip_runtime.h"
#ifdef DEBUG
#include <stdio.h>
#endif
#ifndef __NVCC__
#include <stdlib.h>
#include <assert.h>
#include <omp.h>
#endif
#include <stdint.h>
#include "filter.h"

static const int dimx = 12, dimy = dimx; 

#define minInt(a,b) (((a)<(b))?(a):(b))
#define maxInt(a,b) (((a)<(b))?(b):(a))


#ifdef __NVCC__
__global__
void cuda_box_blur(const uint8_t __restrict__ *src, uint8_t __restrict__ *dest, const int height, const int width){
    // const int index = threadIdx.x; const int stride = blockDim.x;
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if(i > width) return;
#else
void box_blur(const uint8_t *restrict src, uint8_t *restrict dest, const int height, const int width){
    // #pragma omp parallel for
    for(int i = 0;i < width;++i){
#endif
        int32_t tmp[3] = {0,0,0};

        for(int i1 = maxInt(0,i-(dimx)/2);i1 < minInt(i+(dimx+1)/2,width);++i1){
            for(int j1 = 0;j1 < minInt((dimy-1)/2,height);++j1){
                for(int ch = 0; ch < 3;++ch){
                    tmp[ch] += src[3*(height*i1+j1)+ch];
                }
            }
        }
        for(int j = 0;j < height;++j){
            
            const int j1 = j-(dimy+1)/2,
                      j2 = j+(dimy-1)/2;
            for(int i1 = maxInt(0,i-(dimx-1)/2);i1 < minInt(i+(dimx+1)/2,width);++i1){
                for(int ch = 0; ch < 3;++ch){
                    tmp[ch] -= (j1 >= 0    )?src[3*(height*i1+j1)+ch]:0;
                    tmp[ch] += (j2 < height)?src[3*(height*i1+j2)+ch]:0;
                }
            }
            const int cnt = (minInt(i+((dimx+1)/2),width)-maxInt(0,i-((dimx-1)/2)))*(minInt(j+((dimy+1)/2),height)-maxInt(0,j-(dimy-1)/2));

            for(int ch = 0; ch < 3;++ch){
                dest[3*(height*i+j)+ch] = (uint8_t)(tmp[ch]/cnt);
            }
        }
#ifndef __NVCC__
    } // for
#endif
}

#ifdef __NVCC__
const int NTHREADS = 512;
extern "C" {
void box_blur(const uint8_t * src, uint8_t * dest, const int height, const int width){
	cuda_box_blur<<<(width+NTHREADS-1)/NTHREADS,NTHREADS>>>(src,  dest, height, width);
}
}
#endif
